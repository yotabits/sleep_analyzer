#include "hip/hip_runtime.h"
/*
 * find_min.cu
 *
 *  Created on: 19 nov. 2015
 *      Author: tkostas
 */

#include "find_min.cuh"
#include "stdio.h"

void cpu_max(unsigned char *values, unsigned int size, unsigned char gpu_max);



void __global__ find_min_k(unsigned char *input, unsigned char *output, unsigned int nb_elt)
{
	extern __shared__ unsigned char sdatac[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdatac[tid] = 0;

	if (i  < nb_elt)
	{
		if (i + blockDim.x > nb_elt - 1)
			sdatac[tid] = input[i * 3];
		else if (i < nb_elt)
			sdatac[tid] = fminf(input[i * 3 + blockDim.x * 3], input[i * 3]);
	}
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
			sdatac[tid] = fminf(sdatac[tid + s], sdatac[tid]);

		__syncthreads();
	}

	if (tid == 0)
		output[blockIdx.x * 3] = sdatac[0];
}

void __global__ find_max_k(unsigned char *input, unsigned char *output, unsigned int nb_elt)
{
	extern __shared__ unsigned char sdatac[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdatac[tid] = 0;

	if (i  < nb_elt)
	{
		if (i + blockDim.x > nb_elt - 1)
			sdatac[tid] = input[i * 3];
		else if (i < nb_elt)
			sdatac[tid] = fmaxf(input[i * 3 + blockDim.x * 3], input[i * 3]);
	}
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
			sdatac[tid] = fmaxf(sdatac[tid + s], sdatac[tid]);

		__syncthreads();
	}

	if (tid == 0)
		output[blockIdx.x * 3] = sdatac[0];
}


unsigned char find_min(unsigned int pixel_size, unsigned char *input, unsigned char *output, unsigned char *temp_input) // min value is located in output[0]
{

	unsigned int threads;
	unsigned int block;
	get_threads_blocks(&threads, &block, pixel_size);

	if (block % 2 == 0)
		block = block / 2;
	else
		block = block / 2 + 1;

	bool run_loop = true;
	bool first_kernel_run = true;

	while (run_loop)
	{
		if (block == 1)
			run_loop = false;
		if (first_kernel_run)
			find_min_k <<<block, threads, threads * sizeof(unsigned char) >>>(input, output, pixel_size);
		else
			find_min_k <<<block, threads, threads * sizeof(unsigned char) >>>(temp_input, output, pixel_size);
		first_kernel_run = false;
		hipDeviceSynchronize();
		hipMemcpy(temp_input, output, block * sizeof(unsigned char) * 3, hipMemcpyDeviceToDevice);
		pixel_size = block;
		get_threads_blocks(&threads, &block, pixel_size);
		if (block % 2 == 0)
			block = block / 2;
		else
			block = block / 2 + 1;
	}
	unsigned char min_found;
	printf("ERROR += %s \n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(&min_found, output, sizeof(unsigned char), hipMemcpyDeviceToHost);
	//cpu_max(input,1000000 , min_found);
	return min_found;
}


unsigned char find_max(unsigned int pixel_size, unsigned char *input, unsigned char *output, unsigned char *temp_input) // min value is located in output[0]
{

	unsigned int threads;
	unsigned int block;
	get_threads_blocks(&threads, &block, pixel_size);

	if (block % 2 == 0)
		block = block / 2;
	else
		block = block / 2 + 1;

	bool run_loop = true;
	bool first_kernel_run = true;

	while (run_loop)
	{
		if (block == 1)
			run_loop = false;
		if (first_kernel_run)
			find_max_k <<<block, threads, threads * sizeof(unsigned char) >>>(input, output, pixel_size);
		else
			find_max_k <<<block, threads, threads * sizeof(unsigned char) >>>(temp_input, output, pixel_size);
		first_kernel_run = false;
		hipDeviceSynchronize();
		hipMemcpy(temp_input, output, block * sizeof(unsigned char) * 3, hipMemcpyDeviceToDevice);
		pixel_size = block;
		get_threads_blocks(&threads, &block, pixel_size);
		if (block % 2 == 0)
			block = block / 2;
		else
			block = block / 2 + 1;
	}
	unsigned char max_found;
	printf("ERROR += %s \n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(&max_found, output, sizeof(unsigned char), hipMemcpyDeviceToHost);
	//cpu_max(input,1000000 , min_found);
	return max_found;
}
/////////////////////////DEBUG FUNCTIONS/////////////////////////////////////////////////////


void cpu_max(unsigned char *values, unsigned int size, unsigned char gpu_max)
{
	unsigned char *values_cpu = (unsigned char*) malloc(sizeof(unsigned char) * size * 3);
	hipMemcpy(values_cpu, values, size * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	unsigned char max = 0;
	for (int i = 0; i < size; i++)
	{

		if (max <  values_cpu[i * 3])
		{
			max = values_cpu[i * 3];
			printf("cpu val = %i \n", values_cpu[i * 3] );
		}
	}

	if (gpu_max != max)
	{
	printf("cpu_max = %i %i\n", max, gpu_max);
	getchar();
	}
	free(values_cpu);
}

void make_random(float **vector, unsigned int vec_size)
{
	float *v1_gpu_x;
	hipMalloc(&v1_gpu_x, sizeof(float)* vec_size);
	float *v1_cpu_x = (float*)malloc(sizeof(float)* vec_size);
	for (unsigned int i = 0; i < vec_size; i++)
	{
		v1_cpu_x[i] = (float)(vec_size - i);
	}
	//cpu_min(v1_cpu_x, vec_size);
	hipMemcpy(v1_gpu_x, v1_cpu_x, sizeof(float)* vec_size, hipMemcpyHostToDevice);
	free(v1_cpu_x);
	*vector = v1_gpu_x;
}
